#include "hip/hip_runtime.h"
#include <iostream>


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "../CUDA_Practice-master/include/matrix.cuh"




int getThreadNum(int gpu = 0)

{

	hipDeviceProp_t prop;

	int count, maxThreadsPerBlock = 0;



	HANDLE_ERROR(hipGetDeviceCount(&count));

	printf("GPU num: %d\n", count);



	for (size_t i = 0; i < count; i++)

	{

		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

		printf("GPU: %d\n", i);

		printf("Max thread num: %d\n", prop.maxThreadsPerBlock);

		printf("Max grid dimensions: %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);



		if (gpu == i)

		{

			maxThreadsPerBlock = prop.maxThreadsPerBlock;

		}

	}



	return maxThreadsPerBlock;

}



__global__ void

conv(float* imgGPU, float* kernelGPU, float* resultGPU,

	int width, int height, int kernelSize)

{

	int thIdx = threadIdx.x;

	int blkIdx = blockIdx.x;

	int id = blkIdx * blockDim.x + thIdx;

	if (id < width * height)

	{

		int row = id / width, // ���������к�

			col = id % width; // ���������к�



		for (int i = 0; i < kernelSize; ++i)

		{

			for (int j = 0; j < kernelSize; ++j)

			{

				float val = 0.0;

				int curRow = row - kernelSize / 2 + i,

					curCol = col - kernelSize / 2 + j;



				if (curRow >= 0 && curCol >= 0 && curRow < height && curCol < width)

					val = imgGPU[curRow * width + curCol];



				resultGPU[id] += kernelGPU[i * kernelSize + j] * val;
				//resultGPU[id] ++;
				
			}

		}

	}
	
}





int main(int argc, char const* argv[])

{

	int width = 1920, height = 1080;

	// float *img = new float[width * height];



	Matrix<float> img(width, height), result(width, height);

	for (size_t row = 0; row < height; ++row)

	{

		for (size_t col = 0; col < width; ++col)

		{

			img(row, col) = (row + col) % 256;

		}

	}

	img.printData();



	int kernelSize = 3;

	Matrix<float> kernel(kernelSize);

	for (size_t index = 0; index < kernel.getLength(); ++index)

	{

		kernel.data[index] = float(index % kernelSize) - 1.0;

	}

	kernel.printData();



	// GPU data

	float* imgGPU = img.toCUDA();

	float* kernelGPU = kernel.toCUDA();

	float* resultGPU;

	HANDLE_ERROR(hipMalloc((void**)&resultGPU, img.getLength() * sizeof(float)));



	int threadNum = getThreadNum();

	int blockNum = (img.getLength() - 0.5) / threadNum + 1;

	conv << <blockNum, threadNum >> > (imgGPU, kernelGPU, resultGPU, width, height, kernelSize);

	// conv<<<blockNum, width*height>>>(imgGPU, kernelGPU, resultGPU, width, height, kernelSize);



	result.toCPU(resultGPU);

	result.printData();



	return 0;

}