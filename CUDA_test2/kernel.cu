
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;//����ʹ��1��block�����̣߳�������Ҫʹ��thredid �ο���p31ҳ
    c[i] = a[i] + b[i];
}

int main() {
	int width = 1920;
	int height = 1080;
	float* img = new float[width * height];
	for (int row = 0; row < height; row++)
	{
		for (int col = 0; col < width; col++) {
			img[row * width + col] = (col + row) % 256;
		}
	}
	int kernelSize = 3;
	float* kernel = new float[kernelSize * kernelSize];

	for (int i = 0; i < kernelSize * kernelSize; i++)
	{
		kernel[i] = i % kernelSize - 1;
	}

	//visualization
	for (int row = 0; row < 10; row++)
	{
		for (int col = 0; col < 10; col++) {
			printf("%2.0f", img[row * width + col]);

		}
		printf("\n");
	}

	return 0;
}