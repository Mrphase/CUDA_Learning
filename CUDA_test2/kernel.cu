
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;//由于使用1个block，个线程，所以需要使用thredid 参考书p31页
    c[i] = a[i] + b[i];
}

int main() {
	int width = 1920;
	int height = 1080;
	float* img = new float[width * height];
	for (int row = 0; row < height; row++)
	{
		for (int col = 0; col < width; col++) {
			img[row * width + col] = (col + row) % 256;
		}
	}
	int kernelSize = 3;
	float* kernel = new float[kernelSize * kernelSize];

	for (int i = 0; i < kernelSize * kernelSize; i++)
	{
		kernel[i] = i % kernelSize - 1;
	}

	//visualization
	for (int row = 0; row < 10; row++)
	{
		for (int col = 0; col < 10; col++) {
			printf("%2.0f", img[row * width + col]);

		}
		printf("\n");
	}

	return 0;
}