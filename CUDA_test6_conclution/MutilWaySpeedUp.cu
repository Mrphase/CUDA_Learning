#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "malloc.h"
#include "gputimer.h"
const int N = 16; //size of mat

void Transpose_Normal(int in[], int out[]){
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			out[i * N + j] = in[j * N + i];
		}
	}

}

__global__ void Transpose_GPU_ROW(int in[], int out[]) {
	int id = threadIdx.x;
	for (int i = 0; i < N; i++)
	{
		out[i * N + id] = in[id * N + i];  //?????????  data hazard?

	}
}

__global__ void Transpose_GPU_Element(int in[], int out[]) {
	int idx = blockIdx.x*16 + threadIdx.x;  //16 thread each block
	int idy = blockIdx.y*16+threadIdx.y;
	out[idx * N + idy] = in[idy * N + idx];
}

__global__ void Transpose_GPU_Element2(int in[], int out[]) {
	//??????????????????????use any way to define idx and idy dose matter?  Transpose 1,2 is correctboth and time using is simmilar
	int idx =threadIdx.x;  //16 thread each block    //use any way to define idx and idy dose matter?  Transpose 1,2 is correctboth and time using is simmilar
	int idy = blockIdx.x;
	out[idx * N + idy] = in[idy * N + idx];
}


int K = 32;

__global__ void transpose_parallel_per_element_tiled(float in[], float out[])
{
	int in_corner_i = blockIdx.x * K, in_corner_j = blockIdx.y * K;
	int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

	int x = threadIdx.x, y = threadIdx.y;

	__shared__ float tile[K][K];

	tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y) * N];
	__syncthreads();
	out[(out_corner_i + x) + (out_corner_j + y) * N] = title[x][y];
}

void Print_Matrix(int in[]) {
	printf("below is %c\n", in);
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%d  ", in[i * N + j]);
		}
		printf("\n");
	}
	printf("finish����������������������������������������������������\n");
}
int main(){

	int sizeOfMat = N * N * sizeof(int);

	int* in = (int*) malloc(sizeOfMat);
	int* out = (int*)malloc(sizeOfMat);
	int* gold = (int*)malloc(sizeOfMat);
	int temp = 0;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++) {
			in[i * N + j] = temp;
			gold[j * N + i] = temp;
			temp++;
		}
	}
	////GPU
	int* GPU_in, * GPU_out;
	hipMalloc(&GPU_in, sizeOfMat);
	hipMalloc(&GPU_out, sizeOfMat);
	hipMemcpy(GPU_in, in, sizeOfMat, hipMemcpyHostToDevice);

	////Gpu- Row
	GpuTimer timer;  //???? how to judge auglothim improvement runing on different device?  
	//such as: someone write A program running in 1 second, i change his' and running in 0.5 second on differint device, how can we compare this program?

	//
	timer.Start();
	Transpose_GPU_ROW << <1, N >> > (GPU_in, GPU_out);
	timer.Stop();
	printf("Time Transpose_GPU_ROW  = %g ms\n", timer.Elapsed()); // ���


	////Gpu- Element
	GpuTimer timer2;
	timer2.Start();
	dim3 block(N / 16, N / 16);
	dim3 thread(16, 16);
	//Transpose_GPU_Element << <block, thread >> > (GPU_in, GPU_out);
	Transpose_GPU_Element2 << <block, thread >> > (GPU_in, GPU_out);
	timer2.Stop();
	printf("Time Transpose_GPU_Element  = %g ms\n", timer2.Elapsed()); // ���


	Print_Matrix(in);
	//Print_Matrix(gold);
	
	Transpose_Normal(in, out);
	Print_Matrix(out);
	return 0;

}