#include "hip/hip_runtime.h"
//C++
#include <time.h>
#include <iostream>
using namespace std;


#include "hip/hip_runtime.h"
#include ""
#include "../CUDA_test6_conclution/gputimer.h"
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <cmath>
void Print_Matrix(int in[]) {
	printf("below is %c\n", in);
	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			printf("%d  ", in[i * 10 + j]);
		}
		printf("\n");
	}
	printf("finish����������������������������������������������������\n");
}
// �˺���
__global__ void transformKernel(float* output,
	hipTextureObject_t texObj,
	int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x<0 || x>width || y<0 || y>height)
		return;
	output[x * width + y] = tex2D<float>(texObj, x + 0.5f, y + 0.5f);
}

__global__ void Transpose_GPU_Element(int in[], int out[]) {
	int idx = threadIdx.x;  //16 thread each block    //use any way to define idx and idy dose matter?  Transpose 1,2 is correctboth and time using is simmilar
	int idy = blockIdx.x;
	out[idx * 1024 + idy] = in[idy * 1024 + idx];
}
int main()
{
	
	int width = 1024;
	int height = 1024;
	int size = width * height * sizeof(float);

	float* h_data = new float[width * height];

	for (int y = 0; y < height; y++)
	{
		for (int x = 0; x < width; x++)
		{
			h_data[y * width + x] = x;
		}
	}
	printf("origin \n");
	for (int y = 0; y < 10; y++)
	{
		for (int x = 0; x < 10; x++)
		{
			printf("%2.0f ", h_data[y * width + x]);
		}
		printf("\n");
	}


	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);
	hipMemcpyToArray(cuArray, 0, 0, h_data, size, hipMemcpyHostToDevice);

	// �����������
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	float* output;
	hipMalloc((void**)&output, size);

	// ���ú˺���
	dim3 dimBlock(4, 4);
	dim3 dimGrid(max((width + dimBlock.x - 1) / dimBlock.x, 1),
		max((height + dimBlock.y - 1) / dimBlock.y, 1));



	///////////////////////////////////////////////////copy from test6
	int N = 1024;
	int sizeOfMat = N * N * sizeof(int);

	int* in = (int*)malloc(sizeOfMat);
	int* out = (int*)malloc(sizeOfMat);
	int* gold = (int*)malloc(sizeOfMat);
	int temp = 0;
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++) {
			in[i * N + j] = j;
			//printf("<%2.0f,%2.0f>",i, h_data[i * width + j]);
			gold[j * N + i] = j;
			//temp++;

		}
	}
	int* GPU_in, * GPU_out;
	hipMalloc(&GPU_in, sizeOfMat);
	hipMalloc(&GPU_out, sizeOfMat);
	hipMemcpy(GPU_in, h_data, sizeOfMat, hipMemcpyHostToDevice);
	////Gpu- Element
	GpuTimer timer2;
	timer2.Start();
	dim3 block(N / 16, N / 16);
	dim3 thread(16, 16);
	Transpose_GPU_Element << <N, N >> > (GPU_in, GPU_out);

	hipMemcpy(out, GPU_out,sizeOfMat, hipMemcpyDeviceToHost);
	//Transpose_GPU_Element2 << <block, thread >> > (GPU_in, GPU_out

	timer2.Stop();
	printf("Time Transpose_GPU_Element  = %g ms\n", timer2.Elapsed()); // ���
	printf("Element \n", out);
	hipFree(GPU_out);

	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			printf("%d ", gold[i * 1024 + j]);
		}
		printf("\n");
	}
	printf("finish����������������������������������������������������\n");
	///////////////////////////////////////////////////copy from test6


	//Gpu- texture
	GpuTimer timer;
	timer.Start();
	transformKernel << <dimGrid, dimBlock >> > (output,
		texObj,
		width, height);;
	timer.Stop();
	printf("Time Transpose_Using_texture  = %g ms\n", timer.Elapsed()); // ���




	hipMemcpy(h_data, output, size, hipMemcpyDeviceToHost);
	printf("texture  \n");
	for (int y = 0; y < 10; y++)
	{
		for (int x = 0; x < 10; x++)
		{
			printf("%2.0f ", h_data[y * 1024 + x]);
		}
		printf("\n");
	}

	// �����������
	hipDestroyTextureObject(texObj);

	// �ͷ��豸�ڴ�
	hipFreeArray(cuArray);
	hipFree(output);

	delete[]h_data;

	return 0;
}

