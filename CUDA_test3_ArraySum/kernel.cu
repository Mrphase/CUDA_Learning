
#include "hip/hip_runtime.h"


#include <stdio.h>
#define _CRT_SECURE_NO_WARNINGS
#define _CRT_SECURE_NO_WARNINGS
__global__ void sum(float a, float b) {
	int id = threadIdx.x;

	//__shared__ float sdata[16];


}
int main()
{
	float a[16];
	for (int i = 0; i < 16; i++)

	{
		a[i] = i * (i + 1);

	}
	float* aGpu;
	hipMalloc((void**)&aGpu, 16 * sizeof(float));
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

	float* bGpu;
	hipMalloc((void**)&bGpu, 1 * sizeof(float));
	sum <<<1, 16>>> (*aGpu, *bGpu);

	float b[1];
	hipMemcpy(bGpu, b, 1 * sizeof(float), hipMemcpyDeviceToHost);

	printf("b: %f\n", b);

    return 0;
}
