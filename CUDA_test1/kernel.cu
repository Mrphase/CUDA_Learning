
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include <stdio.h>
//# define num 10 
__global__ void add1(int* a, int* b, int* c, int nu) {
	int i = threadIdx.x;
	if (i < nu) {
		c[i] = b[i] + a[i];
		//__syncthreads();
	}
	//__syncthreads();
}
int main(void) {
	const int num = 10;//没有从上图报错 ： 表达式必须有常量值
	int a[num], b[10], c[10];
	int* a_gpu, * b_gpu, * c_gpu;

	for (int i = 0; i < num; i++) {
		a[i] = i;
		b[i] = i * i;

	}
	hipMalloc((void**)&a_gpu, num * sizeof(int));
	hipMalloc((void**)&b_gpu, num * sizeof(int));
	hipMalloc((void**)&c_gpu, num * sizeof(int));

	//copy data
	hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

	
	add1 <<<1,10>>>(a_gpu, b_gpu, c_gpu, num); //应输入表达式，解决： 应输入表达式： 形参出问题， 是*a 不是a 与文件名无关

	//将此处改为<<<10,1>>>会出现问题： c为0
	//get data 
	hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

	//visualization
	for (int i = 0; i < num; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c [i]);
	}

	return 0;


}