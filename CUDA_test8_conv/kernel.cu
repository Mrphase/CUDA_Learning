#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

int getNum() {
	hipDeviceProp_t prop;
	int num;
	hipGetDeviceCount(&num);
	hipGetDeviceProperties(&prop, 0);
	printf("thread num = %d\n", prop.maxThreadsPerBlock);
	printf("thread num = %d , %d,%d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	//printf("thread num = %d", prop.maxThreadsPerBlock);
	return prop.maxThreadsPerBlock;
}

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;//����ʹ��1��block�����̣߳�������Ҫʹ��thredid �ο���p31ҳ
	c[i] = a[i] + b[i];
}

__global__ void conv(float *img, float *kernel,
	float *result, int width, int height, int kernelSize) {
	int ti = threadIdx.x;
	int bi = blockIdx.x;
	int id = (bi * blockDim.x+ti);// 1024 is thread num
	//int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id >= width*height)
	{
		return;
	}
	int row = id / width;
	int col = id % width;

	for (int i = 0; i < 3; ++i)
	{
		for (int j = 0; j < 3; ++j) {
			float imgValue = 0.0;
			int curRow = row - kernelSize /2 +i;
			int curCol = col - kernelSize / 2 + j;
			
			//if (curCol < 0 || curRow < 0 || curCol >= width || curRow >= height) {}

			if (curRow >= 0 && curCol >= 0 && curRow < height && curCol < width)
			{
				imgValue = img[curRow * width + curCol];
				
				//printf("%2.0f", result[id]);
				result[id] += kernel[i * kernelSize + j] * imgValue;
			}
			
			//printf("!!!!!!!!%2.0f", result[id]);
			//result[id] += kernel[i * kernelSize + j] * imgValue;
		}
		
	}
	//printf("!!!!!!!!%2.0f", result[0]);
}

int main() {
	int width = 1920;
	int height = 1080;
	float* img = new float[width * height];
	for (int row = 0; row < height; row++)
	{
		for (int col = 0; col < width; col++) {
			img[row * width + col] = (col + row) % 256;
		}
	}
	int kernelSize = 3;
	float* kernel = new float[kernelSize * kernelSize];

	for (int i = 0; i < kernelSize * kernelSize; i++)
	{
		kernel[i] = i % kernelSize - 1;
	}

	float *d_img;
	float *d_kernel;
	float *d_result;

	hipMalloc((void**)&d_img, width * height * sizeof(float));
	hipMalloc((void**)&d_kernel, kernelSize * kernelSize* sizeof(float)); //3*3
	hipMalloc((void**)&d_result, width * height * sizeof(float));
	
	hipMemcpy(d_img,img, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, img, 9 * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy((void**)&d_result, img, width * height * sizeof(float), hipMemcpyHostToDevice);

	//need num of block and thread
	int threadNum =  getNum();//thread num
	int blockNum = (width * height - 0.5) / threadNum + 1;

	conv << <blockNum, threadNum >> > 
		(d_img, d_kernel, d_result, width, height, kernelSize);
	
	float* result = new float[width * height];
	hipMemcpy(result, d_result,
		width * height * sizeof(float), hipMemcpyDeviceToHost);


	//visualization
	printf("img\n");
	for (int row = 0; row < 10; row++)
	{
		for (int col = 0; col < 10; col++) {
			//printf("%2.0f��%2.0f ___", row, col);
			printf("%2.0f ", img[col + row * width]);

		}
		printf("\n");
	}
	//printf("kernel\n");
	printf("kernel\n");
	for (int row = 0; row < kernelSize; row++)
	{
		for (int col = 0; col < kernelSize; col++) {
			printf("%2.0f ", kernel[col + row * kernelSize]);

		}
		printf("\n");
	}

	printf("result\n");
	for (int row = 0; row < 5; row++)
	{
		for (int col = 0; col < 5; col++) {
			//printf("%2.0f��%2.0f ___", row,col);
			printf("%2.0f ", result[col + row * width]);

		}
		printf("\n");
	}
	printf("result: %2.0f",result[0]);




	return 0;
}