
#include "hip/hip_runtime.h"




#include "hip/device_functions.h"
#include <stdio.h>


__global__ void sum11(float *a, float *b) {
	int id = threadIdx.x;

	__shared__ float sdata[16];
	sdata[id] = a[id]; //赋值不用for ,一个线程做一件事
	__syncthreads();

	for (int i = 8; i >0; i/=2)
	{
		if (id < i)
		{

			sdata[id] += sdata[id + i];
		}
		__syncthreads(); //内存同步

	}
	if (id==0)
	{
		b[0] = sdata[0];
	}
}
int main()
{
	float a[16];
	for (int i = 0; i < 16; i++)

	{
		a[i] = i ;

	}
	float* aGpu;
	hipMalloc((void**)&aGpu, 16 * sizeof(float));
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

	float* bGpu;
	hipMalloc((void**)&bGpu, 1 * sizeof(float));
	sum11 <<<1, 16 >>> (aGpu, bGpu);//应输入表达式： 形参出问题， 是*a 不是a

	float b[1];
	hipMemcpy(b, bGpu, 1 * sizeof(float), hipMemcpyDeviceToHost);

	printf("b: %f\n", b[0]);

	return 0;
}
