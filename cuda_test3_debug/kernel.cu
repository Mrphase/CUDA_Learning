
#include "hip/hip_runtime.h"




#include "hip/device_functions.h"
#include <stdio.h>


__global__ void sum11(float *a, float *b) {
	int id = threadIdx.x;

	__shared__ float sdata[16];
	sdata[id] = a[id]; //��ֵ����for ,һ���߳���һ����
	__syncthreads();

	for (int i = 8; i >0; i/=2)
	{
		if (id < i)
		{

			sdata[id] += sdata[id + i];
		}
		__syncthreads(); //�ڴ�ͬ��

	}
	if (id==0)
	{
		b[0] = sdata[0];
	}
}
int main()
{
	float a[16];
	for (int i = 0; i < 16; i++)

	{
		a[i] = i ;

	}
	float* aGpu;
	hipMalloc((void**)&aGpu, 16 * sizeof(float));
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);

	float* bGpu;
	hipMalloc((void**)&bGpu, 1 * sizeof(float));
	sum11 <<<1, 16 >>> (aGpu, bGpu);//Ӧ������ʽ�� �βγ����⣬ ��*a ����a

	float b[1];
	hipMemcpy(b, bGpu, 1 * sizeof(float), hipMemcpyDeviceToHost);

	printf("b: %f\n", b[0]);

	return 0;
}
